#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>                                                                                   
                                                                           
#include <chrono>                                                                                   
#include <iostream> 

#define KERNELS 1000000

// # threadblocks
#define TBLOCKS 32
#define THREADS 16

//__global__ void memcpy_kernel_batch(int *dst, int *src, size_t n)
__global__ void memcpy_kernel_batch()
{
    for (int j = 0; j < 10000000; j++)
    {
        int id = blockDim.x * blockIdx.x + threadIdx.x;
        id++; 
   }
}


int main(int argc, char **argv)
{
    auto start = std::chrono::system_clock::now();
    for (int j = 0; j < KERNELS; j++)
        memcpy_kernel_batch<<<TBLOCKS, THREADS, 0, 0>>>();
    hipDeviceSynchronize();
    auto end = std::chrono::system_clock::now();
    std::chrono::duration<double, std::milli> elapsed_milli = end - start;
    std::cerr<<"Elapsed time: "<<elapsed_milli.count()<<std::endl;
    return 0;
}
